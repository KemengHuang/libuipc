#include <muda/logger.h>
#include <kernel_cout.h>

namespace uipc::backend::cuda
{
__device__ muda::LoggerViewer cout;

muda::LoggerViewer* device_logger_viewer_ptr() noexcept
{
    muda::LoggerViewer* ptr = nullptr;
    // In IDE this will show an error, but it's fine
    // don't change the `cout` to `&cout`
    // or you get runtime error
    checkCudaErrors(hipGetSymbolAddress((void**)&ptr, &cout));
    return ptr;
}
}  // namespace uipc::backend::cuda

#include <contact_system/simplex_normal_contact.h>
#include <contact_system/contact_models/codim_ipc_simplex_normal_contact_function.h>
#include <utils/distance/distance_flagged.h>
#include <utils/codim_thickness.h>
#include <kernel_cout.h>

namespace uipc::backend::cuda
{
class IPCSimplexNormalContact final : public SimplexNormalContact
{
  public:
    using SimplexNormalContact::SimplexNormalContact;

    virtual void do_build(BuildInfo& info) override
    {
        auto constitution =
            world().scene().info()["contact"]["constitution"].get<std::string>();
        if(constitution != "ipc")
        {
            throw SimSystemException("Constitution is not IPC");
        }
    }

    virtual void do_compute_energy(EnergyInfo& info) override
    {
        using namespace muda;
        using namespace sym::codim_ipc_simplex_contact;

        // Compute Point-Triangle energy
        auto PT_count = info.PTs().size();
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(PT_count,
                   [table = info.contact_tabular().viewer().name("contact_tabular"),
                    contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                    PTs = info.PTs().viewer().name("PTs"),
                    Es  = info.PT_energies().viewer().name("Es"),
                    Ps  = info.positions().viewer().name("Ps"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    d_hat = info.d_hat(),
                    dt    = info.dt()] __device__(int i) mutable
                   {
                       Vector4i PT = PTs(i);

                       Vector4i cids  = {contact_ids(PT[0]),
                                         contact_ids(PT[1]),
                                         contact_ids(PT[2]),
                                         contact_ids(PT[3])};
                       Float    kappa = PT_kappa(table, cids) * dt * dt;

                       const auto& P  = Ps(PT[0]);
                       const auto& T0 = Ps(PT[1]);
                       const auto& T1 = Ps(PT[2]);
                       const auto& T2 = Ps(PT[3]);


                       Float thickness = PT_thickness(thicknesses(PT(0)),
                                                      thicknesses(PT(1)),
                                                      thicknesses(PT(2)),
                                                      thicknesses(PT(3)));

                       Vector4i flag =
                           distance::point_triangle_distance_flag(P, T0, T1, T2);

                       if constexpr(RUNTIME_CHECK)
                       {
                           Float D;
                           distance::point_triangle_distance2(flag, P, T0, T1, T2, D);

                           Vector2 range = D_range(thickness, d_hat);

                           MUDA_ASSERT(is_active_D(range, D),
                                       "PT[%d,%d,%d,%d] d^2(%f) out of range, (%f,%f)",
                                       PT(0),
                                       PT(1),
                                       PT(2),
                                       PT(3),
                                       D,
                                       range(0),
                                       range(1));
                       }

                       Es(i) = PT_barrier_energy(flag, kappa, d_hat, thickness, P, T0, T1, T2);
                   });

        // Compute Edge-Edge energy
        auto EE_count = info.EEs().size();
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(EE_count,
                   [table = info.contact_tabular().viewer().name("contact_tabular"),
                    contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                    EEs = info.EEs().viewer().name("EEs"),
                    Es  = info.EE_energies().viewer().name("Es"),
                    Ps  = info.positions().viewer().name("Ps"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                    d_hat   = info.d_hat(),
                    dt      = info.dt()] __device__(int i) mutable
                   {
                       Vector4i EE = EEs(i);

                       Vector4i cids  = {contact_ids(EE[0]),
                                         contact_ids(EE[1]),
                                         contact_ids(EE[2]),
                                         contact_ids(EE[3])};
                       Float    kappa = EE_kappa(table, cids) * dt * dt;

                       const auto& E0 = Ps(EE[0]);
                       const auto& E1 = Ps(EE[1]);
                       const auto& E2 = Ps(EE[2]);
                       const auto& E3 = Ps(EE[3]);

                       const auto& t0_Ea0 = rest_Ps(EE[0]);
                       const auto& t0_Ea1 = rest_Ps(EE[1]);
                       const auto& t0_Eb0 = rest_Ps(EE[2]);
                       const auto& t0_Eb1 = rest_Ps(EE[3]);

                       Float thickness = EE_thickness(thicknesses(EE(0)),
                                                      thicknesses(EE(1)),
                                                      thicknesses(EE(2)),
                                                      thicknesses(EE(3)));

                       Vector4i flag = distance::edge_edge_distance_flag(E0, E1, E2, E3);

                       if constexpr(RUNTIME_CHECK)
                       {
                           Float D;
                           distance::edge_edge_distance2(flag, E0, E1, E2, E3, D);
                           Vector2 range = D_range(thickness, d_hat);

                           MUDA_ASSERT(is_active_D(range, D),
                                       "EE[%d,%d,%d,%d] d^2(%f) out of range, (%f,%f), [%d,%d,%d,%d]",
                                       EE(0),
                                       EE(1),
                                       EE(2),
                                       EE(3),
                                       D,
                                       range(0),
                                       range(1),
                                       flag(0),
                                       flag(1),
                                       flag(2),
                                       flag(3));
                       }


                       Es(i) = mollified_EE_barrier_energy(flag,
                                                           // coefficients
                                                           kappa,
                                                           d_hat,
                                                           thickness,
                                                           // positions
                                                           t0_Ea0,
                                                           t0_Ea1,
                                                           t0_Eb0,
                                                           t0_Eb1,
                                                           E0,
                                                           E1,
                                                           E2,
                                                           E3);
                   });

        // Compute Point-Edge energy
        auto PE_count = info.PEs().size();
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(PE_count,
                   [table = info.contact_tabular().viewer().name("contact_tabular"),
                    contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                    PEs     = info.PEs().viewer().name("PEs"),
                    Es      = info.PE_energies().viewer().name("Es"),
                    Ps      = info.positions().viewer().name("Ps"),
                    rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    eps_v = info.eps_velocity(),
                    d_hat = info.d_hat(),
                    dt    = info.dt()] __device__(int i) mutable
                   {
                       Vector3i PE = PEs(i);

                       Vector3i cids  = {contact_ids(PE[0]),
                                         contact_ids(PE[1]),
                                         contact_ids(PE[2])};
                       Float    kappa = PE_kappa(table, cids) * dt * dt;

                       const auto& P  = Ps(PE[0]);
                       const auto& E0 = Ps(PE[1]);
                       const auto& E1 = Ps(PE[2]);

                       Float thickness = PE_thickness(thicknesses(PE(0)),
                                                      thicknesses(PE(1)),
                                                      thicknesses(PE(2)));

                       Vector3i flag = distance::point_edge_distance_flag(P, E0, E1);

                       if constexpr(RUNTIME_CHECK)
                       {
                           Float D;
                           distance::point_edge_distance2(flag, P, E0, E1, D);

                           Vector2 range = D_range(thickness, d_hat);

                           MUDA_ASSERT(is_active_D(range, D),
                                       "PE[%d,%d,%d] d^2(%f) out of range, (%f,%f)",
                                       PE(0),
                                       PE(1),
                                       PE(2),
                                       D,
                                       range(0),
                                       range(1));
                       }

                       Es(i) = PE_barrier_energy(flag, kappa, d_hat, thickness, P, E0, E1);
                   });

        // Compute Point-Point energy
        auto PP_count = info.PPs().size();
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(PP_count,
                   [table = info.contact_tabular().viewer().name("contact_tabular"),
                    contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                    PPs     = info.PPs().viewer().name("PPs"),
                    Es      = info.PP_energies().viewer().name("Es"),
                    Ps      = info.positions().viewer().name("Ps"),
                    rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    d_hat = info.d_hat(),
                    dt    = info.dt()] __device__(int i) mutable
                   {
                       Vector2i PP = PPs(i);

                       Vector2i cids = {contact_ids(PP[0]), contact_ids(PP[1])};
                       Float    kappa = PP_kappa(table, cids) * dt * dt;

                       const auto& Pa = Ps(PP[0]);
                       const auto& Pb = Ps(PP[1]);

                       Float thickness =
                           PP_thickness(thicknesses(PP(0)), thicknesses(PP(1)));

                       Vector2i flag = distance::point_point_distance_flag(Pa, Pb);

                       if constexpr(RUNTIME_CHECK)
                       {
                           Float D;
                           distance::point_point_distance2(flag, Pa, Pb, D);

                           Vector2 range = D_range(thickness, d_hat);

                           MUDA_ASSERT(is_active_D(range, D),
                                       "PP[%d,%d] d^2(%f) out of range, (%f,%f)",
                                       PP(0),
                                       PP(1),
                                       D,
                                       range(0),
                                       range(1));
                       }

                       Es(i) = PP_barrier_energy(flag, kappa, d_hat, thickness, Pa, Pb);
                   });
    }

    virtual void do_assemble(ContactInfo& info) override
    {
        using namespace muda;
        using namespace sym::codim_ipc_simplex_contact;

        if(info.PTs().size())
        {
            // Compute Point-Triangle Gradient and Hessian
            ParallelFor()
                .file_line(__FILE__, __LINE__)
                .apply(info.PTs().size(),
                       [table = info.contact_tabular().viewer().name("contact_tabular"),
                        contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                        PTs = info.PTs().viewer().name("PTs"),
                        Gs  = info.PT_gradients().viewer().name("Gs"),
                        Hs  = info.PT_hessians().viewer().name("Hs"),
                        Ps  = info.positions().viewer().name("Ps"),
                        thicknesses = info.thicknesses().viewer().name("thicknesses"),
                        d_hat = info.d_hat(),
                        dt    = info.dt()] __device__(int i) mutable
                       {
                           Vector4i PT = PTs(i);

                           Vector4i cids  = {contact_ids(PT[0]),
                                             contact_ids(PT[1]),
                                             contact_ids(PT[2]),
                                             contact_ids(PT[3])};
                           Float    kappa = PT_kappa(table, cids) * dt * dt;


                           const auto& P  = Ps(PT[0]);
                           const auto& T0 = Ps(PT[1]);
                           const auto& T1 = Ps(PT[2]);
                           const auto& T2 = Ps(PT[3]);


                           Float thickness = PT_thickness(thicknesses(PT(0)),
                                                          thicknesses(PT(1)),
                                                          thicknesses(PT(2)),
                                                          thicknesses(PT(3)));

                           Vector4i flag =
                               distance::point_triangle_distance_flag(P, T0, T1, T2);

                           if constexpr(RUNTIME_CHECK)
                           {
                               Float D;
                               distance::point_triangle_distance2(flag, P, T0, T1, T2, D);

                               Vector2 range = D_range(thickness, d_hat);

                               MUDA_ASSERT(is_active_D(range, D),
                                           "PT[%d,%d,%d,%d] d^2(%f) out of range, (%f,%f)",
                                           PT(0),
                                           PT(1),
                                           PT(2),
                                           PT(3),
                                           D,
                                           range(0),
                                           range(1));
                           }

                           PT_barrier_gradient_hessian(
                               Gs(i), Hs(i), flag, kappa, d_hat, thickness, P, T0, T1, T2);

                           //cout << "Gs: " << Gs(i).transpose().eval() << "\n"
                           //     << "Hs: " << Hs(i).transpose().eval() << "\n";
                       });
        }


        // Compute Edge-Edge Gradient and Hessian
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(info.EEs().size(),
                   [table = info.contact_tabular().viewer().name("contact_tabular"),
                    contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                    EEs = info.EEs().viewer().name("EEs"),
                    Gs  = info.EE_gradients().viewer().name("Gs"),
                    Hs  = info.EE_hessians().viewer().name("Hs"),
                    Ps  = info.positions().viewer().name("Ps"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                    d_hat   = info.d_hat(),
                    dt      = info.dt()] __device__(int i) mutable
                   {
                       Vector4i EE = EEs(i);

                       Vector4i cids  = {contact_ids(EE[0]),
                                         contact_ids(EE[1]),
                                         contact_ids(EE[2]),
                                         contact_ids(EE[3])};
                       Float    kappa = EE_kappa(table, cids) * dt * dt;

                       const auto& E0 = Ps(EE[0]);
                       const auto& E1 = Ps(EE[1]);
                       const auto& E2 = Ps(EE[2]);
                       const auto& E3 = Ps(EE[3]);

                       const auto& t0_Ea0 = rest_Ps(EE[0]);
                       const auto& t0_Ea1 = rest_Ps(EE[1]);
                       const auto& t0_Eb0 = rest_Ps(EE[2]);
                       const auto& t0_Eb1 = rest_Ps(EE[3]);

                       Float thickness = EE_thickness(thicknesses(EE(0)),
                                                      thicknesses(EE(1)),
                                                      thicknesses(EE(2)),
                                                      thicknesses(EE(3)));

                       Vector4i flag = distance::edge_edge_distance_flag(E0, E1, E2, E3);

                       if constexpr(RUNTIME_CHECK)
                       {
                           Float D;
                           distance::edge_edge_distance2(flag, E0, E1, E2, E3, D);

                           Vector2 range = D_range(thickness, d_hat);

                           MUDA_ASSERT(is_active_D(range, D),
                                       "EE[%d,%d,%d,%d] d^2(%f) out of range, (%f,%f)",
                                       EE(0),
                                       EE(1),
                                       EE(2),
                                       EE(3),
                                       D,
                                       range(0),
                                       range(1));
                       }

                       mollified_EE_barrier_gradient_hessian(
                           Gs(i), Hs(i), flag, kappa, d_hat, thickness, t0_Ea0, t0_Ea1, t0_Eb0, t0_Eb1, E0, E1, E2, E3);
                   });

        // Compute Point-Edge Gradient and Hessian
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(info.PEs().size(),
                   [table = info.contact_tabular().viewer().name("contact_tabular"),
                    contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                    PEs     = info.PEs().viewer().name("PEs"),
                    Gs      = info.PE_gradients().viewer().name("Gs"),
                    Hs      = info.PE_hessians().viewer().name("Hs"),
                    Ps      = info.positions().viewer().name("Ps"),
                    rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    d_hat = info.d_hat(),
                    dt    = info.dt()] __device__(int i) mutable
                   {
                       Vector3i PE = PEs(i);

                       Vector3i cids  = {contact_ids(PE[0]),
                                         contact_ids(PE[1]),
                                         contact_ids(PE[2])};
                       Float    kappa = PE_kappa(table, cids) * dt * dt;

                       const auto& P  = Ps(PE[0]);
                       const auto& E0 = Ps(PE[1]);
                       const auto& E1 = Ps(PE[2]);

                       Float thickness = PE_thickness(thicknesses(PE(0)),
                                                      thicknesses(PE(1)),
                                                      thicknesses(PE(2)));

                       Vector3i flag = distance::point_edge_distance_flag(P, E0, E1);

                       if constexpr(RUNTIME_CHECK)
                       {
                           Float D;
                           distance::point_edge_distance2(flag, P, E0, E1, D);

                           Vector2 range = D_range(thickness, d_hat);

                           MUDA_ASSERT(is_active_D(range, D),
                                       "PE[%d,%d,%d] d^2(%f) out of range, (%f,%f)",
                                       PE(0),
                                       PE(1),
                                       PE(2),
                                       D,
                                       range(0),
                                       range(1));
                       }

                       PE_barrier_gradient_hessian(
                           Gs(i), Hs(i), flag, kappa, d_hat, thickness, P, E0, E1);
                   });

        // Compute Point-Point Gradient and Hessian
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(info.PPs().size(),
                   [table = info.contact_tabular().viewer().name("contact_tabular"),
                    contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                    PPs = info.PPs().viewer().name("PPs"),
                    Gs  = info.PP_gradients().viewer().name("Gs"),
                    Hs  = info.PP_hessians().viewer().name("Hs"),
                    Ps  = info.positions().viewer().name("Ps"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    d_hat = info.d_hat(),
                    dt    = info.dt()] __device__(int i) mutable
                   {
                       const auto& PP = PPs(i);

                       Vector2i cids = {contact_ids(PP[0]), contact_ids(PP[1])};
                       Float    kappa = PP_kappa(table, cids) * dt * dt;

                       const auto& P0 = Ps(PP[0]);
                       const auto& P1 = Ps(PP[1]);

                       Float thickness =
                           PP_thickness(thicknesses(PP(0)), thicknesses(PP(1)));

                       Vector2i flag = distance::point_point_distance_flag(P0, P1);

                       if constexpr(RUNTIME_CHECK)
                       {
                           Float D;
                           distance::point_point_distance2(flag, P0, P1, D);

                           Vector2 range = D_range(thickness, d_hat);

                           MUDA_ASSERT(is_active_D(range, D),
                                       "PP[%d,%d] d^2(%f) out of range, (%f,%f)",
                                       PP(0),
                                       PP(1),
                                       D,
                                       range(0),
                                       range(1));
                       }

                       PP_barrier_gradient_hessian(
                           Gs(i), Hs(i), flag, kappa, d_hat, thickness, P0, P1);
                   });
    }
};

REGISTER_SIM_SYSTEM(IPCSimplexNormalContact);
}  // namespace uipc::backend::cuda
#include <contact_system/simplex_normal_contact.h>
#include <muda/ext/eigen/evd.h>
#include <muda/cub/device/device_merge_sort.h>
#include <utils/distance.h>
#include <utils/codim_thickness.h>

namespace uipc::backend::cuda
{
void SimplexNormalContact::do_build()
{
    m_impl.global_trajectory_filter = &require<GlobalTrajectoryFilter>();
    m_impl.global_contact_manager   = &require<GlobalContactManager>();
    m_impl.global_vertex_manager    = &require<GlobalVertexManager>();

    BuildInfo info;
    do_build(info);

    m_impl.global_contact_manager->add_reporter(this);
    m_impl.dt = world().scene().info()["dt"].get<Float>();

    on_init_scene(
        [this]
        {
            m_impl.simplex_trajectory_filter =
                m_impl.global_trajectory_filter->find<SimplexTrajectoryFilter>();
        });
}

void SimplexNormalContact::Impl::compute_energy(SimplexNormalContact* contact,
                                                GlobalContactManager::EnergyInfo& info)
{
    EnergyInfo this_info{this};

    auto filter = simplex_trajectory_filter;

    PT_count = filter->PTs().size();
    EE_count = filter->EEs().size();
    PE_count = filter->PEs().size();
    PP_count = filter->PPs().size();

    auto count_4 = (PT_count + EE_count);
    auto count_3 = PE_count;
    auto count_2 = PP_count;

    energies.resize(count_4 + count_3 + count_2);

    SizeT offset            = 0;
    this_info.m_PT_energies = energies.view(offset, PT_count);
    offset += PT_count;
    this_info.m_EE_energies = energies.view(offset, EE_count);
    offset += EE_count;
    this_info.m_PE_energies = energies.view(offset, PE_count);
    offset += PE_count;
    this_info.m_PP_energies = energies.view(offset, PP_count);


    contact->do_compute_energy(this_info);
    using namespace muda;

    // if(info.is_initial())
    //{
    //    DeviceMergeSort().SortKeys(energies.data(),
    //                               energies.size(),
    //                               [] HIPCUB_RUNTIME_FUNCTION(Float a, Float b)
    //                               { return a < b; });
    //}

    DeviceReduce().Sum(energies.data(), info.energy().data(), energies.size());
}

void SimplexNormalContact::do_compute_energy(GlobalContactManager::EnergyInfo& info)
{
    m_impl.compute_energy(this, info);
}

void SimplexNormalContact::do_report_extent(GlobalContactManager::ContactExtentInfo& info)
{
    auto& filter = m_impl.simplex_trajectory_filter;

    // m_impl.classify_constraints();
    //m_impl.PT_count = m_impl.PT_constraints.size();
    //m_impl.EE_count = m_impl.EE_constraints.size();
    //m_impl.PE_count = m_impl.PE_constraints.size();
    //m_impl.PP_count = m_impl.PP_constraints.size();

    m_impl.PT_count = filter->PTs().size();
    m_impl.EE_count = filter->EEs().size();
    m_impl.PE_count = filter->PEs().size();
    m_impl.PP_count = filter->PPs().size();


    auto count_4 = (m_impl.PT_count + m_impl.EE_count);
    auto count_3 = m_impl.PE_count;
    auto count_2 = m_impl.PP_count;

    // expand to hessian3x3 and graident3
    SizeT contact_gradient_count = 4 * count_4 + 3 * count_3 + 2 * count_2;
    SizeT contact_hessian_count = 4 * 4 * count_4 + 3 * 3 * count_3 + 2 * 2 * count_2;

    info.gradient_count(contact_gradient_count);
    info.hessian_count(contact_hessian_count);

    m_impl.loose_resize(m_impl.PT_EE_gradients, count_4);
    m_impl.loose_resize(m_impl.PT_EE_hessians, count_4);

    m_impl.loose_resize(m_impl.PE_gradients, count_3);
    m_impl.loose_resize(m_impl.PE_hessians, count_3);

    m_impl.loose_resize(m_impl.PP_gradients, count_2);
    m_impl.loose_resize(m_impl.PP_hessians, count_2);
}

void SimplexNormalContact::do_assemble(GlobalContactManager::ContactInfo& info)
{
    ContactInfo this_info{&m_impl};

    this_info.m_PT_gradients = m_impl.PT_EE_gradients.view(0, m_impl.PT_count);
    this_info.m_EE_gradients =
        m_impl.PT_EE_gradients.view(m_impl.PT_count, m_impl.EE_count);
    this_info.m_PE_gradients = m_impl.PE_gradients.view();
    this_info.m_PP_gradients = m_impl.PP_gradients.view();

    this_info.m_PT_hessians = m_impl.PT_EE_hessians.view(0, m_impl.PT_count);
    this_info.m_EE_hessians = m_impl.PT_EE_hessians.view(m_impl.PT_count, m_impl.EE_count);
    this_info.m_PE_hessians = m_impl.PE_hessians.view();
    this_info.m_PP_hessians = m_impl.PP_hessians.view();

    // let subclass to fill in the data
    do_assemble(this_info);

    // _assemble the data to the global contact manager
    m_impl.assemble(info);
}

muda::CBuffer2DView<ContactCoeff> SimplexNormalContact::BaseInfo::contact_tabular() const
{
    return m_impl->global_contact_manager->contact_tabular();
}

muda::CBufferView<Vector4i> SimplexNormalContact::BaseInfo::PTs() const
{
    // return m_impl->PT_constraints.view();
    return m_impl->simplex_trajectory_filter->PTs();
}

muda::CBufferView<Vector4i> SimplexNormalContact::BaseInfo::EEs() const
{
    return m_impl->simplex_trajectory_filter->EEs();
}

muda::CBufferView<Vector3i> SimplexNormalContact::BaseInfo::PEs() const
{
    return m_impl->simplex_trajectory_filter->PEs();
}

muda::CBufferView<Vector2i> SimplexNormalContact::BaseInfo::PPs() const
{
    return m_impl->simplex_trajectory_filter->PPs();
}

muda::CBufferView<Float> SimplexNormalContact::BaseInfo::thicknesses() const
{
    return m_impl->global_vertex_manager->thicknesses();
}

muda::CBufferView<Vector3> SimplexNormalContact::BaseInfo::positions() const
{
    return m_impl->global_vertex_manager->positions();
}

muda::CBufferView<Vector3> SimplexNormalContact::BaseInfo::prev_positions() const
{
    return m_impl->global_vertex_manager->prev_positions();
}

muda::CBufferView<Vector3> SimplexNormalContact::BaseInfo::rest_positions() const
{
    return m_impl->global_vertex_manager->rest_positions();
}

muda::CBufferView<IndexT> SimplexNormalContact::BaseInfo::contact_element_ids() const
{
    return m_impl->global_vertex_manager->contact_element_ids();
}

Float SimplexNormalContact::BaseInfo::d_hat() const
{
    return m_impl->global_contact_manager->d_hat();
}

Float SimplexNormalContact::BaseInfo::dt() const
{
    return m_impl->dt;
}

Float SimplexNormalContact::BaseInfo::eps_velocity() const
{
    return m_impl->global_contact_manager->eps_velocity();
}

namespace detail
{
    template <SizeT N>
    __inline__ __device__ void fill_contact_hessian(muda::TripletMatrixViewer<Float, 3>& H3x3,
                                                    int                      I,
                                                    const Vector<IndexT, N>& D,
                                                    const Matrix<Float, 3 * N, 3 * N>& H)
    {
        auto offset = N * N * I;
#pragma unroll
        for(int i = 0; i < N; ++i)
        {
            auto L = D(i);
#pragma unroll
            for(int j = 0; j < N; ++j)
            {
                auto R = D(j);
                H3x3(offset++).write(L, R, H.template block<3, 3>(3 * i, 3 * j));
            }
        }
    }

    template <SizeT N>
    __inline__ __device__ void fill_contact_gradient(muda::DoubletVectorViewer<Float, 3>& G3,
                                                     int                      I,
                                                     const Vector<IndexT, N>& D,
                                                     const Vector<Float, 3 * N>& G)
    {
        auto offset = N * I;
#pragma unroll
        for(int i = 0; i < N; ++i)
        {
            auto L = D(i);
            G3(offset++).write(L, G.segment<3>(3 * i));
        }
    }

    template <SizeT N>
    __inline__ __device__ void make_spd(Matrix<Float, N, N>& mat)
    {
        Vector<Float, N>    eigen_values;
        Matrix<Float, N, N> eigen_vectors;
        muda::eigen::template evd(mat, eigen_values, eigen_vectors);
#pragma unroll
        for(int i = 0; i < N; ++i)
        {
            auto& v = eigen_values(i);
            v       = v < 0.0 ? 0.0 : v;
        }
        mat = eigen_vectors * eigen_values.asDiagonal() * eigen_vectors.transpose();
    }
}  // namespace detail


void SimplexNormalContact::Impl::assemble(GlobalContactManager::ContactInfo& info)
{
    using namespace muda;

    auto H3x3 = info.hessian();
    auto G3   = info.gradient();
    auto PTs  = simplex_trajectory_filter->PTs();
    auto EEs  = simplex_trajectory_filter->EEs();
    auto PEs  = simplex_trajectory_filter->PEs();
    auto PPs  = simplex_trajectory_filter->PPs();

    auto PT_hessian  = PT_EE_hessians.view(0, PTs.size());
    auto PT_gradient = PT_EE_gradients.view(0, PTs.size());
    auto EE_hessian  = PT_EE_hessians.view(PTs.size(), EEs.size());
    auto EE_gradient = PT_EE_gradients.view(PTs.size(), EEs.size());

    SizeT H3x3_offset = 0;
    SizeT G3_offset   = 0;

    // PT
    {
        SizeT H3x3_count = PTs.size() * 16;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PT_hessian.size(),
                   [PT_H12x12s = PT_hessian.cviewer().name("H12x12"),
                    PTs        = PTs.cviewer().name("PTs"),
                    H3x3 = H3x3.subview(H3x3_offset, H3x3_count).viewer().name("H3x3")] __device__(int I) mutable
                   {
                       Matrix12x12 H12x12 = PT_H12x12s(I);
                       Vector4i    D4     = PTs(I);
                       detail::make_spd<12>(H12x12);
                       detail::fill_contact_hessian<4>(H3x3, I, D4, H12x12);
                   });
        H3x3_offset += H3x3_count;

        SizeT G3_count = PT_gradient.size() * 4;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PT_gradient.size(),
                   [PT_G12s = PT_gradient.cviewer().name("G12"),
                    PTs     = PTs.cviewer().name("PTs"),
                    G3 = G3.subview(G3_offset, G3_count).viewer().name("G3")] __device__(int I) mutable
                   {
                       const auto& G12 = PT_G12s(I);
                       Vector4i    D4  = PTs(I);
                       detail::fill_contact_gradient<4>(G3, I, D4, G12);
                   });
        G3_offset += G3_count;
    }

    // EE
    {
        SizeT H3x3_count = EEs.size() * 16;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(EE_hessian.size(),
                   [EE_H12x12s = EE_hessian.cviewer().name("H12x12"),
                    EEs        = EEs.cviewer().name("EEs"),
                    H3x3 = H3x3.subview(H3x3_offset, H3x3_count).viewer().name("H3x3")] __device__(int I) mutable
                   {
                       Matrix12x12 H12x12 = EE_H12x12s(I);
                       Vector4i    D4     = EEs(I);
                       detail::make_spd<12>(H12x12);
                       detail::fill_contact_hessian<4>(H3x3, I, D4, H12x12);
                   });

        H3x3_offset += H3x3_count;

        SizeT G3_count = EE_gradient.size() * 4;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(EE_gradient.size(),
                   [G12s = EE_gradient.cviewer().name("G12"),
                    EEs  = EEs.cviewer().name("EEs"),
                    G3 = G3.subview(G3_offset, G3_count).viewer().name("G3")] __device__(int I) mutable
                   {
                       const auto& G12 = G12s(I);
                       Vector4i    D4  = EEs(I);
                       detail::fill_contact_gradient<4>(G3, I, D4, G12);
                   });

        G3_offset += G3_count;
    }

    // PE
    {
        SizeT H3x3_count = PEs.size() * 9;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PEs.size(),
                   [PE_H9x9s = PE_hessians.cviewer().name("H9x9"),
                    PEs      = PEs.cviewer().name("PEs"),
                    H3x3 = H3x3.subview(H3x3_offset, H3x3_count).viewer().name("H3x3")] __device__(int I) mutable
                   {
                       Matrix9x9 H9x9 = PE_H9x9s(I);
                       Vector3i  D3   = PEs(I);
                       detail::make_spd<9>(H9x9);
                       detail::fill_contact_hessian<3>(H3x3, I, D3, H9x9);
                   });

        H3x3_offset += H3x3_count;

        SizeT G3_count = PEs.size() * 3;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PEs.size(),
                   [G9s = PE_gradients.cviewer().name("G9"),
                    PEs = PEs.cviewer().name("PEs"),
                    G3 = G3.subview(G3_offset, G3_count).viewer().name("G3")] __device__(int I) mutable
                   {
                       const auto& G9 = G9s(I);
                       Vector3i    D3 = PEs(I);
                       detail::fill_contact_gradient<3>(G3, I, D3, G9);
                   });

        G3_offset += G3_count;
    }


    // PP
    {
        SizeT H3x3_count = PPs.size() * 4;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PPs.size(),
                   [PP_H6x6s = PP_hessians.cviewer().name("H6x6"),
                    PPs      = PPs.cviewer().name("PPs"),
                    H3x3 = H3x3.subview(H3x3_offset, H3x3_count).viewer().name("H3x3")] __device__(int I) mutable
                   {
                       Matrix6x6 H6x6 = PP_H6x6s(I);
                       Vector2i  D2   = PPs(I);
                       detail::make_spd<6>(H6x6);
                       detail::fill_contact_hessian<2>(H3x3, I, D2, H6x6);
                   });

        H3x3_offset += H3x3_count;

        SizeT G3_count = PPs.size() * 2;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PPs.size(),
                   [G6s = PP_gradients.cviewer().name("G6"),
                    PPs = PPs.cviewer().name("PPs"),
                    G3 = G3.subview(G3_offset, G3_count).viewer().name("G3")] __device__(int I) mutable
                   {
                       const auto& G6 = G6s(I);
                       Vector2i    D2 = PPs(I);
                       detail::fill_contact_gradient<2>(G3, I, D2, G6);
                   });

        G3_offset += G3_count;
    }

    UIPC_ASSERT(H3x3_offset == info.hessian().triplet_count(), "size mismatch");
    UIPC_ASSERT(G3_offset == info.gradient().doublet_count(), "size mismatch");
}
}  // namespace uipc::backend::cuda
